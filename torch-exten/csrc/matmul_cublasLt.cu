#include <hipblaslt.h>
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#define CUBLAS_RETURN_IF_ERROR(expr) \
  do {                               \
    hipblasStatus_t status = (expr);  \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
      if (status == HIPBLAS_STATUS_INVALID_VALUE) \
        std::cout << "call " << #expr << "failed, ret: invalid value" << std::endl; \
      else \
        std::cout << "call " << #expr << "failed, ret: " << status << std::endl; \
      return; \
    } \
  } while (0)

namespace torch_ext {
template <typename T, hipblasComputeType_t COMPUTE_TYPE, hipDataType DATA_TYPE>
void cublasLtMatmulHelper(
    hipblasLtHandle_t lthandle,
    bool transa,
    bool transb,
    int m,
    int n,
    int k,
    float alpha,
    const T *A,
    int lda,
    const T *B,
    int ldb,
    float beta,
    T *C,
    int ldc,
    hipStream_t stream) {
  hipblasLtMatmulDesc_t operationDesc = NULL;
  hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
  hipblasLtMatmulPreference_t preference = NULL;

  int returnedResults = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};

  // create operation desciriptor; see hipblasLtMatmulDescAttributes_t for details about defaults; here we just need to
  // set the transforms for A and B
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescCreate(&operationDesc, COMPUTE_TYPE, HIP_R_32F));
  hipblasOperation_t transA = transa ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t transB = transb ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transA, sizeof(transA)));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transB, sizeof(transB)));

  // create matrix descriptors, we are good with the details here so no need to set any extra attributes
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutCreate(&Adesc, DATA_TYPE, transa ? k : m, transa ? m : k, lda));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutCreate(&Bdesc, DATA_TYPE, transb ? n : k, transb ? k : n, ldb));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutCreate(&Cdesc, DATA_TYPE, m, n, ldc));

  // create preference handle; here we could use extra attributes to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C; here for simplicity we just assume A,B,C are always well aligned (e.g.
  // directly come from hipMalloc)

  // allocate workspace
  uint64_t workspaceSize = 1024*1024*4;
  void* workspace = nullptr;
  // uint32_t align = 1;
  hipMallocAsync(&workspace, workspaceSize, stream);
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulPreferenceCreate(&preference));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize)));
  // CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulPreferenceSetAttribute(preference, CUBLASLT_MATMUL_PREF_MIN_ALIGNMENT_A_BYTES, &align, sizeof(align)));
  // CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulPreferenceSetAttribute(preference, CUBLASLT_MATMUL_PREF_MIN_ALIGNMENT_B_BYTES, &align, sizeof(align)));
  // CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulPreferenceSetAttribute(preference, CUBLASLT_MATMUL_PREF_MIN_ALIGNMENT_C_BYTES, &align, sizeof(align)));
  // CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulPreferenceSetAttribute(preference, CUBLASLT_MATMUL_PREF_MIN_ALIGNMENT_D_BYTES, &align, sizeof(align)));

  // we just need the best available heuristic to try and run matmul. There is no guarantee this will work, e.g. if A
  // is badly aligned, you can request more (e.g. 32) algos and try to run them one by one until something works
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulAlgoGetHeuristic(lthandle, operationDesc, Adesc, Bdesc, Cdesc, Cdesc, preference, 1, &heuristicResult, &returnedResults));

  if (returnedResults == 0) {
    TORCH_CHECK(false, "cublasLt get no results");
  }

  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmul(lthandle,
                                    operationDesc,
                                    &alpha,
                                    A,
                                    Adesc,
                                    B,
                                    Bdesc,
                                    &beta,
                                    C,
                                    Cdesc,
                                    C,
                                    Cdesc,
                                    &heuristicResult.algo,
                                    workspace,
                                    workspaceSize,
                                    stream));
  hipFreeAsync(workspace, stream);

  // descriptors are no longer needed as all GPU work was already enqueued
  if (preference) CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulPreferenceDestroy(preference));
  if (Cdesc) CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutDestroy(Cdesc));
  if (Bdesc) CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutDestroy(Bdesc));
  if (Adesc) CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutDestroy(Adesc));
  if (operationDesc) CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescDestroy(operationDesc));
  return;
}

void hipblasLtMatmul(const torch::Tensor &A, const torch::Tensor &B, torch::Tensor &C) {
  assert (A.dim() == 2 && B.dim() == 2 && C.dim() == 2);
  int M = A.size(0);
  int N = B.size(1);
  int K = A.size(1);
  assert (K == B.size(0) && M == C.size(0) && N == C.size(1));
  hipblasLtHandle_t lthandle;
  CUBLAS_RETURN_IF_ERROR(hipblasLtCreate(&lthandle));

  const at::cuda::OptionalCUDAGuard device_guard(device_of(A));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  if (A.dtype() == at::ScalarType::Float) {
    return cublasLtMatmulHelper<float, HIPBLAS_COMPUTE_32F, HIP_R_32F>(
        lthandle, false, false, N, M, K, 1.0f, B.data_ptr<float>(), N, A.data_ptr<float>(), K, 0.0f, C.data_ptr<float>(), N, stream);
  } else if (A.dtype() == at::ScalarType::Half) {
    return cublasLtMatmulHelper<at::Half, HIPBLAS_COMPUTE_32F, HIP_R_16F>(
        lthandle, false, false, N, M, K, 1.0f, B.data_ptr<at::Half>(), N, A.data_ptr<at::Half>(), K, 0.0f, C.data_ptr<at::Half>(), N, stream);
  } else {
    TORCH_CHECK(false, "Unsupported data type: ", A.dtype());
  }
  CUBLAS_RETURN_IF_ERROR(hipblasLtDestroy(lthandle));
}

}  // namespace torch_ext
